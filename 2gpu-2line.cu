#include "hip/hip_runtime.h"
#include <cstddef>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include "enigma.cu"

//__device__ bool dev_found;
char Alphabet[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R', 'S', 'T', 'U',
               'V', 'W', 'X', 'Y', 'Z'};

void swapTwoElement(char* arr, int index1, int index2){
    char temp = arr[index1];
    arr[index1] = arr[index2];
    arr[index2] = temp;
}

__device__ bool endingPatternCompare(char* text, int textLength, char* pattern, int patternLength){
    int start = textLength - patternLength;
    for(int i = 0; i < patternLength; i++){
        if(pattern[i] != text[start + i]){
            return false;
        }
    }
    return true;
}

__global__ void decrypt(char* inputText, size_t textLength, char* plugBoard, int* rotorCombination, bool* found, int* key){
    int patternLength = 10;
    char pattern[10] = {'H','E','I','L','H','I','T','L','E','R'};
    char startPoint[3];
    int rotorOrder[3];
    
    ///// Set start point
    startPoint[0] = stdAlphabet[blockIdx.x];
    startPoint[1] = stdAlphabet[blockIdx.y];
    startPoint[2] = stdAlphabet[blockIdx.z];

    ///// Select rotors
    rotorOrder[0] = rotorCombination[threadIdx.x * 3];
    rotorOrder[1] = rotorCombination[threadIdx.x * 3 + 1];
    rotorOrder[2] = rotorCombination[threadIdx.x * 3 + 2];
    
    char outputText[100];
    
    int startComparison = textLength - patternLength;
    
    #pragma unroll
    for(int i = 0; i < startComparison; i++){
        rotate(rotorOrder, startPoint);
    }
    enigma_run(inputText + startComparison, outputText, patternLength, plugBoard, rotorOrder, startPoint);

    if(endingPatternCompare(outputText, patternLength, pattern, patternLength)){
        found[0] = true;
        key[0] = rotorOrder[0];
        key[1] = rotorOrder[1];
        key[2] = rotorOrder[2];
        key[3] = blockIdx.x;
        key[4] = blockIdx.y;
        key[5] = blockIdx.z;
    }
}

int main(){
    timespec startTime, endTime;
    clock_gettime(CLOCK_MONOTONIC, &startTime);
    hipError_t status;
    size_t textLength = 15;
    char inputText[15] = {'G','R','A','U','I','D','B','G','M','P','Z','R','K','R','O'};
    char* inputText_dev1;
    char* inputText_dev2;
    char* plugBoard_dev1;
    char* plugBoard_dev2;
    char plugBoard_h[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
    int* rotorCombination_dev1;
    int* rotorCombination_dev2;
    bool found[1];
    found[0] = false;
    bool* found_dev1;
    bool* found_dev2;
    int key[6];
    int* key_dev1;
    int* key_dev2;
    char plugBoardKey[26];
    bool lock = false;

    int rotorCombination[3 * 60];
    int count = 0;
    for(int i = 0; i < 5; i++){
        for(int j = 0; j < 5; j++){
            for(int k = 0; k < 5; k++){
                if(i != j && i != k && j != k){
                    rotorCombination[count * 3] = i;
                    rotorCombination[count * 3 + 1] = j;
                    rotorCombination[count * 3 + 2] = k;
                    count++;
                }
            }
        }
    }

    hipSetDevice(0);
    status = hipMalloc((void**)&inputText_dev1, textLength * sizeof(char));
    status = hipMalloc((void**)&plugBoard_dev1, 26 * sizeof(char));
    status = hipMalloc((void**)&rotorCombination_dev1, 180 * sizeof(int));
    status = hipMemcpy(inputText_dev1, inputText, textLength * sizeof(char), hipMemcpyHostToDevice);
    status = hipMemcpy(plugBoard_dev1, plugBoard_h, 26 * sizeof(char), hipMemcpyHostToDevice);
    status = hipMemcpy(rotorCombination_dev1, rotorCombination, 180 * sizeof(int), hipMemcpyHostToDevice);
    status = hipMalloc((void**)&found_dev1, sizeof(bool));
    hipMemcpy(found_dev1, found, sizeof(bool), hipMemcpyHostToDevice);
    status = hipMalloc((void**)&key_dev1, 6 * sizeof(int));
    status = hipMemcpy(key_dev1, key, 6 * sizeof(int), hipMemcpyHostToDevice);
    
    if(status != hipSuccess){
        exit(5);
    }

    hipSetDevice(1);
    status = hipMalloc((void**)&inputText_dev2, textLength * sizeof(char));
    status = hipMalloc((void**)&plugBoard_dev2, 26 * sizeof(char));
    status = hipMalloc((void**)&rotorCombination_dev2, 180 * sizeof(int));
    status = hipMemcpy(inputText_dev2, inputText, textLength * sizeof(char), hipMemcpyHostToDevice);
    status = hipMemcpy(plugBoard_dev2, plugBoard_h, 26 * sizeof(char), hipMemcpyHostToDevice);
    status = hipMemcpy(rotorCombination_dev2, rotorCombination, 180 * sizeof(int), hipMemcpyHostToDevice);
    status = hipMalloc((void**)&found_dev2, sizeof(bool));
    hipMemcpy(found_dev2, found, sizeof(bool), hipMemcpyHostToDevice);
    status = hipMalloc((void**)&key_dev2, 6 * sizeof(int));
    status = hipMemcpy(key_dev2, key, 6 * sizeof(int), hipMemcpyHostToDevice);
    if(status != hipSuccess){
        exit(5);
    }
    

    dim3 gridSize(26, 26, 26);
    dim3 blockSize(60, 1, 1);

    // char* plugBoard_d;
    // status = hipMalloc((void**)&plugBoard_d, 26 * sizeof(char));




    int threadRank;
    /////////////////////////////////////////////////////////
    ////// Set plugboard
    /////////////////////////////////////////////////////////

#pragma omp parallel private(threadRank) num_threads(2)
    {
    threadRank = omp_get_thread_num();
    printf("current thread: %d\n", threadRank);
    hipSetDevice(threadRank);
    
    if(threadRank == 0){
    //////// Line1 //////////
        for(int a1 = 0; a1 < 7; a1++){
            for(int b1 = a1 + 1; b1 < 26; b1++){
                //////// Line2 //////////
                for(int a2 = 0; a2 < 26; a2++){
                    if(a2 == a1 || a2 == b1) continue;
                    for(int b2 = a2 + 1; b2 < 26; b2++){
                        if(b2 == a1 || b2 == b1) continue;
                        char plugBoard[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
                        swapTwoElement(plugBoard, a1, b1);  // Plug line 1
                        swapTwoElement(plugBoard, a2, b2);  // Plug line 2
                        status = hipMemcpy(plugBoard_dev1, plugBoard, 26 * sizeof(char), hipMemcpyHostToDevice);
                        decrypt<<<gridSize, blockSize>>>(inputText_dev1, textLength, plugBoard_dev1, rotorCombination_dev1, found_dev1, key_dev1);
                        hipMemcpy(found, found_dev1, sizeof(bool), hipMemcpyDeviceToHost);
                        status = hipMemcpy(key, key_dev1, 6 * sizeof(int), hipMemcpyDeviceToHost);
                        if(found[0]&&!lock){
                            lock = true;
                            for(int i = 0; i < 26; i++){
                                plugBoardKey[i] = plugBoard[i];
                            }
                        }            
                    }
                } 
            }
        }
    }
    if(threadRank == 1){
        for(int a1 = 7; a1 < 26; a1++){
            for(int b1 = a1 + 1; b1 < 26; b1++){
                //////// Line2 //////////
                for(int a2 = 0; a2 < 26; a2++){
                    if(a2 == a1 || a2 == b1) continue;
                    for(int b2 = a2 + 1; b2 < 26; b2++){
                        if(b2 == a1 || b2 == b1) continue;
                        char plugBoard[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
                        swapTwoElement(plugBoard, a1, b1);  // Plug line 1
                        swapTwoElement(plugBoard, a2, b2);  // Plug line 2
                        status = hipMemcpy(plugBoard_dev2, plugBoard, 26 * sizeof(char), hipMemcpyHostToDevice);
                        decrypt<<<gridSize, blockSize>>>(inputText_dev2, textLength, plugBoard_dev2, rotorCombination_dev2, found_dev2, key_dev2);
                        hipMemcpy(found, found_dev2, sizeof(bool), hipMemcpyDeviceToHost);
                        status = hipMemcpy(key, key_dev2, 6 * sizeof(int), hipMemcpyDeviceToHost);
                        if(found[0] && !lock){
                            lock = true;
                            for(int i = 0; i < 26; i++){
                                plugBoardKey[i] = plugBoard[i];
                            }   
                        }
                    }
                }
            }
        }
    }
    }

    if(found[0]){
        printf("Found rotor order: ");
        for(int i = 0; i < 3; i++){
            printf("%d", key[i]);
        }
        printf(" ");
        printf("start point: ");
        for(int i = 3; i < 6; i++){
            printf("%c", Alphabet[key[i]]);
        }
        printf("\nPlugboard: ");
        for(int i = 0; i < 26; i++){
            printf("%c ", plugBoardKey[i]);
        }
        printf("\n");
        
    }
    else{
        printf("not found\n");
    }
    clock_gettime(CLOCK_MONOTONIC, &endTime);
    printf("Total time: %d\n", endTime.tv_sec - startTime.tv_sec);

    return 0;
}