#include "hip/hip_runtime.h"
#include <cstddef>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "enigma.cu"

void swapTwoElement(char* arr, int index1, int index2){
    char temp = arr[index1];
    arr[index1] = arr[index2];
    arr[index2] = temp;
}

__device__ bool endingPatternCompare(char* text, int textLength, char* pattern, int patternLength){
    int start = textLength - patternLength;
    for(int i = 0; i < patternLength; i++){
        if(pattern[i] != text[start + i]){
            return false;
        }
    }
    return true;
}

__global__ void decrypt(char* inputText, size_t textLength, char* plugBoard, int* rotorCombination, bool* found){
    // printf("111");
    // if(threadIdx.x == threadIdx.y || threadIdx.x == threadIdx.z || threadIdx.y == threadIdx.z){
    //     return;
    // }
    
    char pattern[10] = {'H','E','I','L','H','I','T','L','E','R'};
    char startPoint[3];
    int rotorOrder[3];
    
    ///// Set start point
    startPoint[0] = stdAlphabet[blockIdx.x];
    startPoint[1] = stdAlphabet[blockIdx.y];
    startPoint[2] = stdAlphabet[blockIdx.z];

    ///// Select rotors
    rotorOrder[0] = rotorCombination[threadIdx.x * 3];
    rotorOrder[1] = rotorCombination[threadIdx.x * 3 + 1];
    rotorOrder[2] = rotorCombination[threadIdx.x * 3 + 2];
    
    char outputText[15];
    
    enigma_run(inputText, outputText, textLength, plugBoard, rotorOrder, startPoint);
    // bool found = endingPatternCompare(outputText, textLength, pattern, 10);
    if(endingPatternCompare(outputText, textLength, pattern, 10)){
        // printf("Found:StartPoint %c, %c, %c, RotorOrder %d, %d, %d\n", 
        //         startPoint[0], startPoint[1], startPoint[2], rotorOrder[0],rotorOrder[1], rotorOrder[2]);
        // *found = true;
    }
}

int main(){
    timespec startTime, endTime;
    clock_gettime(CLOCK_MONOTONIC, &startTime);
    hipError_t status;
    size_t textLength = 15;
    char inputText[15] = {'L','U','Q','N','H','X','V','L','X','E','H','O','R','L','X'};
    char* inputText_d;
    int* rotorCombination_d;


    status = hipMalloc((void**)&inputText_d, textLength * sizeof(char));
    status = hipMemcpy(inputText_d, inputText, textLength * sizeof(char), hipMemcpyHostToDevice);
    if(status != hipSuccess){
        exit(4);
    }

    dim3 gridSize(26, 26, 26);
    dim3 blockSize(60, 1, 1);

    char* plugBoard_d;
    status = hipMalloc((void**)&plugBoard_d, 26 * sizeof(char));

    // char plugBoard_h[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
    // status = hipMemcpy(plugBoard_d, plugBoard_h, 26 * sizeof(char), hipMemcpyHostToDevice);
    // if(status != hipSuccess){
    //     exit(5);
    // }

    int rotorCombination[3 * 60];
    int count = 0;
    for(int i = 0; i < 5; i++){
        for(int j = 0; j < 5; j++){
            for(int k = 0; k < 5; k++){
                if(i != j && i != k && j != k){
                    rotorCombination[count * 3] = i;
                    rotorCombination[count * 3 + 1] = j;
                    rotorCombination[count * 3 + 2] = k;
                    count++;
                }
            }
        }
    }
    status = hipMalloc((void**)&rotorCombination_d, 180 * sizeof(int));
    status = hipMemcpy(rotorCombination_d, rotorCombination, 180 * sizeof(int), hipMemcpyHostToDevice);
    
    // for(int i = 0; i < 60; i++){
    //     printf("%d %d %d\n", rotorCombination[i*3], rotorCombination[i*3+1], rotorCombination[i*3+2]);
    // }
    // printf("here\n");
    // decrypt<<<gridSize, blockSize>>>(inputText_d, textLength, plugBoard_d);
    /////////////////////////////////////////////////////////
    ////// Set plugboard
    /////////////////////////////////////////////////////////
    bool found[1];
    found[1] = false;
    bool* found_d;
    status = hipMalloc((void**)&found_d, sizeof(bool));
    hipMemcpy(found_d, found, sizeof(bool), hipMemcpyHostToDevice);
    //////// Line1 //////////
    for(int a1 = 0; a1 < 26; a1++){
        for(int b1 = a1 + 1; b1 < 26; b1++){
            //////// Line2 //////////
            for(int a2 = 0; a2 < 26; a2++){
                if(a2 == a1 || a2 == b1) continue;
                for(int b2 = a2 + 1; b2 < 26; b2++){
                    if(b2 == a1 || b2 == b1) continue;
                    char plugBoard_h[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
                    swapTwoElement(plugBoard_h, a1, b1);  // Plug line 1
                    swapTwoElement(plugBoard_h, a2, b2);  // Plug line 2
                    status = hipMemcpy(plugBoard_d, plugBoard_h, 26 * sizeof(char), hipMemcpyHostToDevice);
                    decrypt<<<gridSize, blockSize>>>(inputText_d, textLength, plugBoard_d, rotorCombination_d, found_d);
                    // hipMemcpyFromSymbol(&found, HIP_SYMBOL(&dev_found), sizeof(bool), 0, hipMemcpyDeviceToHost);
                    // hipMemcpy(found, found_d, sizeof(bool), hipMemcpyDeviceToHost);
                }
            }
            
        }
    }
    if(found[0]){
         // plugboard: %d %d, %d %d, %d %d\n", a1, b1, a2, b2, a3, b3);
        printf("Found\n");
    }
    else{
        printf("not found\n");
    }
    clock_gettime(CLOCK_MONOTONIC, &endTime);
    printf("Total time: %d\n", endTime.tv_sec - startTime.tv_sec);

    return 0;
}